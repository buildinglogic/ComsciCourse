
#include <hip/hip_runtime.h>
/* ****************************************
*
* CUDA Kernel: matrix minus 
*
*/


/* ****************************************
 *
 * sub2ind - Column-major indexing of 2D arrays
 *
 */
template <typename T>
__device__ __forceinline__ T sub2ind( T i, T j, T height ) {

  return (i + height*j);

}  // end function 'sub2ind'


/* ****************************************
 *
 *  core kernel
 *
 */

__global__ void apply_filter(double * Iedit, 
	const double * ILaplace,
	const double * B,
	const int * Mask,
	const int m, 
	const int n,
    const int p){

/* thread indices */
        const int j = blockIdx.y*blockDim.y+threadIdx.y;
        const int i = blockIdx.x*blockDim.x+threadIdx.x;
        
/* matrix calculation */
	if ((i >= m) || (j >= n*p) || (Mask[sub2ind(i,j,m)]==0) ){
                return;
        }
        
        Iedit[sub2ind(i,j,m)] = (ILaplace[sub2ind(i,j,m)]-B[sub2ind(i,j,m)])/4.0;
        return ;
}

